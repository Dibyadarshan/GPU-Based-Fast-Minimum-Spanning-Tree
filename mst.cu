#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <ctime>
using namespace std;

int main(){

    freopen("graph.txt", "r", stdin);

    int nodes, edges;
    cin>>nodes>>edges;

    vector<vector<pair<int,int> > > adjacency_list(nodes); 
    
    for(int i = 0; i < edges; ++i){
        int node1, node2, weight;
        cin>>node1>>node2>>weight;
        
        adjacency_list[node1].push_back(make_pair(node2, weight));
        adjacency_list[node2].push_back(make_pair(node1, weight));
    }

    int * V = new int[nodes];
    int * E = new int[2 * edges];
    int * W = new int[2 * edges];

    int cumulative_sum = 0, limit;

    for(int i = 0; i < nodes; ++i){
        V[i] = cumulative_sum;
        limit = adjacency_list[i].size();
        for(int j = 0; j < limit; ++j){
            E[cumulative_sum + j] = adjacency_list[i][j].first;
            W[cumulative_sum + j] = adjacency_list[i][j].second;
        }
        cumulative_sum += limit;
    }

    // for(int i = 0; i < nodes; i++)
    // {
    //     cout<<V[i]<<" ";
    // }
    // cout<<endl;
    // for(int i = 0; i < 2 * edges; i++)
    // {
    //     cout<<E[i]<<" "<<W[i]<<"\n";
    // }

    long long int ans = 0;
    int current = 0;
    int count = 0;

    int *parent = new int[nodes];
    int *weights = new int[nodes];
    bool *inMST = new bool[nodes];

    parent[0] = -1;
    for(int i = 0; i < nodes; ++i) {
        weights[i] = INT_MAX;
        inMST[i] = false;
    }

    clock_t begin = clock();

    thrust::device_vector<int> device_weights(weights, weights + nodes);
    thrust::device_ptr<int> ptr = device_weights.data();

    while(count < nodes-1){
        ++count;
        inMST[current] = true;

        for(int i = 0; i < adjacency_list[current].size(); ++i) {
            int incoming_vertex = adjacency_list[current][i].first;
            if(!inMST[incoming_vertex]) {
                if(device_weights[incoming_vertex] > adjacency_list[current][i].second) {
                    device_weights[incoming_vertex] = adjacency_list[current][i].second;
                    parent[incoming_vertex] = current;
                }
            }
        }

        int min_index = thrust::min_element(ptr, ptr + nodes) - ptr;
        cout<<"Min Weight Index: "<<min_index<<endl;
        
        parent[min_index] = current;
        ans += device_weights[min_index];
        device_weights[min_index] = INT_MAX;
        current = min_index;
    }
    
    clock_t end = clock();

    cout<<"Answer: "<<ans<<endl;

    for(int i = 0; i < nodes; ++i) {
        cout<<i<<"'s parent is "<<parent[i]<<endl;
    }

    double elapsed_time = double(end - begin) / CLOCKS_PER_SEC;
    cout<<"Execution time: "<<elapsed_time<<endl;

    return 0;
}




/*
9 14
0 1 4
0 7 8
1 7 11
1 2 8
2 8 2
2 3 7
2 5 4
7 8 7
7 6 1
6 8 6
6 5 2
3 5 14
3 4 9
4 5 10
*/