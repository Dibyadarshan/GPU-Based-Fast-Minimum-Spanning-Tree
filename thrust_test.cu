#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
using namespace std; 

int main() {
    thrust::device_vector<int> a(1000, 0);
    for (int i = 0; i < 1000; i++) {
        a[i] = 10*i-90;
        cout<<i<<" "<<a[i]<<"\n";
    }
    thrust::device_ptr<int> ptr = a.data();
    cout<<thrust::min_element(ptr, ptr + 1000) - ptr;
    return 0;
}

