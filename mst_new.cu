#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/device_free.h>
#include <ctime>
using namespace std;

__global__ void weightUpdate(int *d_V, int *d_E, int *d_W, int *d_C, int * d_parent, int *d_weights, int *d_inMST) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= (d_V[d_C+1] - d_V[d_C]))
        return;
    int index = d_V[d_C] + id;
    int incoming_vertex = d_E[index];
    int edge_weight = d_W[d_V[index];
    if (d_weights[incoming_vertex] > edge_weight) {
        d_weights[incoming_vertex] = edge_weight;
        d_parent[incoming_vertex] = d_C;
    }
}
 
int main(){

    //freopen("graph.txt", "r", stdin);

    int nodes, edges;
    cin>>nodes>>edges;

    vector<vector<pair<int,int> > > adjacency_list(nodes);     
    for(int i = 0; i < edges; ++i){
        int node1, node2, weight;
        cin>>node1>>node2>>weight;
        
        adjacency_list[node1].push_back(make_pair(node2, weight));
        adjacency_list[node2].push_back(make_pair(node1, weight));
    }

    int * V = new int[nodes+1];
    int * E = new int[2 * edges];
    int * W = new int[2 * edges];

    int cumulative_sum = 0, limit;

    for(int i = 0; i < nodes; ++i){
        V[i] = cumulative_sum;
        limit = adjacency_list[i].size();
        for(int j = 0; j < limit; ++j){
            E[cumulative_sum + j] = adjacency_list[i][j].first;
            W[cumulative_sum + j] = adjacency_list[i][j].second;
        }
        cumulative_sum += limit;
    }
    V[nodes] = 2*edges;
    
    int *d_V, *d_E, *d_W;
    hipMalloc((void **)&d_V, (nodes+1) * sizeof(int));
    hipMalloc((void **)&d_E, 2 * edges * sizeof(int));
    hipMalloc((void **)&d_W, 2 * edges * sizeof(bool));
    hipMemcpy(d_V, V, nodes * sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(d_E, E, 2 * edges * sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, 2 * edges * sizeof(int),  hipMemcpyHostToDevice);

    // for(int i = 0; i < nodes; i++)
    // {
    //     cout<<V[i]<<" ";
    // }
    // cout<<endl;
    // for(int i = 0; i < 2 * edges; i++)
    // {
    //     cout<<E[i]<<" "<<W[i]<<"\n";
    // }

    long long int ans = 0;
    int current = 0;
    int count = 0;

    int *parent = new int[nodes];
    int *weights = new int[nodes];
    bool *inMST = new bool[nodes];

    parent[0] = -1;
    for(int i = 0; i < nodes; ++i) {
        weights[i] = INT_MAX;
        inMST[i] = false;
    }

    int *d_parent, *d_weights, *d_inMST;
    hipMalloc((void **)&d_parent, nodes * sizeof(int));
    hipMalloc((void **)&d_weights, nodes * sizeof(int));
    hipMalloc((void **)&d_inMST, nodes * sizeof(bool));

    
    hipMemcpy(d_parent, parent, nodes * sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, nodes * sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(d_inMST, inMST, nodes * sizeof(bool),  hipMemcpyHostToDevice);

    int C = 0;
    int *d_C;
    hipMemcpy(d_C, &C, sizeof(int), hipMemcpyHostToDevice);

    thrust::device_vector<int> device_weights(weights, weights + nodes);
    thrust::device_ptr<int> ptr = device_weights.data();

    clock_t begin = clock();
    // while all nodes are added to MST
    while(count < nodes-1){
        ++count;
        inMST[current] = true;


        // Find the mininum index
        int min_index = thrust::min_element(ptr, ptr + nodes) - ptr;
        // cout<<"Min Weight Index: "<<min_index<<endl;
        
        // update         
        parent[min_index] = current;
        ans += device_weights[min_index];
        device_weights[min_index] = INT_MAX;
        current = min_index;
    }
    clock_t end = clock();

    // print the parent
    for(int i = 0; i < nodes; ++i) {
        cout<<i<<"'s parent is "<<parent[i]<<endl;
    }
    // sum of all weights
    cout<<"Answer: "<<ans<<endl;

    // print the time
    double elapsed_time = double(end - begin) / CLOCKS_PER_SEC;
    cout<<"Execution time: "<<elapsed_time<<endl;

    // free all memory
    free(V); free(E); free(W);
    free(parent); free(weights); free(inMST); 

    return 0;
}

/*
9 14
0 1 4
0 7 8
1 7 11
1 2 8
2 8 2
2 3 7
2 5 4
7 8 7
7 6 1
6 8 6
6 5 2
3 5 14
3 4 9
4 5 10
*/