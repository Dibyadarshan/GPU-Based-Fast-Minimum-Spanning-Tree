#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/device_free.h>
#include <ctime>
using namespace std;

int main(){

    //freopen("graph.txt", "r", stdin);

    // ======================== Input and Adj list formation ====================================
    // Input nodes and edges
    int nodes, edges;
    cin>>nodes>>edges;

    // create the adjancency list
    vector<vector<pair<int,int> > > adjacency_list(nodes); 
    for(int i = 0; i < edges; ++i){
        int node1, node2, weight;
        cin>>node1>>node2>>weight;
        adjacency_list[node1].push_back(make_pair(node2, weight));
        adjacency_list[node2].push_back(make_pair(node1, weight));
    }

    // create compressed adjancency list
    int * V = new int[nodes];
    int * E = new int[2 * edges];
    int * W = new int[2 * edges];
    int cumulative_sum = 0, limit;
    for(int i = 0; i < nodes; ++i){
        V[i] = cumulative_sum;
        limit = adjacency_list[i].size();
        for(int j = 0; j < limit; ++j){
            E[cumulative_sum + j] = adjacency_list[i][j].first;
            W[cumulative_sum + j] = adjacency_list[i][j].second;
        }
        cumulative_sum += limit;
    }
    // Check 
    // for(int i = 0; i < nodes; i++)
    // {
    //     cout<<V[i]<<" ";
    // }
    // cout<<endl;
    // for(int i = 0; i < 2 * edges; i++)
    // {
    //     cout<<E[i]<<" "<<W[i]<<"\n";
    // }


    // ======================== Variables init ====================================
    // sum of edge weights in MST 
    long long int edge_sum = 0;
    // current vertex under consideration
    int current = 0;
    // count of vertex added to MST
    int count = 0;

    int *parent = new int[nodes];
    vector<int> weights(nodes);
    bool *inMST = new bool[nodes];
    // init parents, weight and inMST array 
    parent[0] = -1;
    for(int i = 0; i < nodes; ++i) {
        weights[i] = INT_MAX;
        inMST[i] = false;
    }

    // device vector for the weights array
    thrust::device_vector<int> device_weights(weights.begin(), weights.end());
    thrust::device_ptr<int> ptr = device_weights.data();


    // ======================== Main code ====================================
    clock_t begin = clock();

    while(count < nodes-1){
        // add current vertex to MST
        ++count;
        inMST[current] = true;

        // update weights and parent arrays as per the current vertex in consideration
        int len = adjacency_list[current].size();
        for(int i = 0; i < len; ++i) {
            int incoming_vertex = adjacency_list[current][i].first;
            if(!inMST[incoming_vertex]) {
                if(weights[incoming_vertex] > adjacency_list[current][i].second) {
                    weights[incoming_vertex] = adjacency_list[current][i].second;
                    parent[incoming_vertex] = current;
                }
            }
        }

        // move/copy the host array to device
        device_weights = weights;
        
        // get the min index
        int min_index = thrust::min_element(ptr, ptr + nodes) - ptr;
        // cout<<"Min Weight Index: "<<min_index<<endl;
        
        // add the least edge weight found outto answer 
        parent[min_index] = current;
        edge_sum += weights[min_index];
        // reset weight to INT_MAX for this vertex as it is already considered in MST
        weights[min_index] = INT_MAX;
        // new current 
        current = min_index;      
    }
    clock_t end = clock();


    // ======================== Results ====================================
    // Print parent of nodes in MST
    // for(int i = 0; i < nodes; ++i) {
    //     cout<<i<<"'s parent is "<<parent[i]<<endl;
    // }
    // Print the sum of edges in MST
    cout<<"Sum of Edges in MST: "<<edge_sum<<endl;

    // Print the time for execution
    double elapsed_time = double(end - begin) / CLOCKS_PER_SEC;
    cout<<"Execution time: "<<elapsed_time<<endl;


    // ======================== Memory Deallocation ====================================
    // thrust::device_free(ptr); 	
    // device_weights.clear();
    // thrust::device_vector<int>().swap(device_weights);
    free(V); free(E); free(W);
    free(parent); free(inMST); 

    return 0;
}


// Sample Input
/*
9 14
0 1 4
0 7 8
1 7 11
1 2 8
2 8 2
2 3 7
2 5 4
7 8 7
7 6 1
6 8 6
6 5 2
3 5 14
3 4 9
4 5 10
*/